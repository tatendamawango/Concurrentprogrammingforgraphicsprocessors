#include "hip/hip_runtime.h"
﻿#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <stdio.h>
#include <cctype>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <atomic>
#include <device_atomic_functions.hpp>

struct Student {
    std::string name;
    int year;
    double grade;
};

std::istream& operator>>(std::istream& is, Student& student) {
    return is >> student.name >> student.year >> student.grade;
}

struct Result {
    std::string name;
    std::string year_grade;
};


__global__ void computeResult(const Student* students, Result* results, int num_students, int* num_results) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < num_students) {

        const Student& student = students[tid];

        char first_letter = std::toupper(student.name[0]);
        if (first_letter > 'P') {
            char grade = getGrade(student);
            
            std::string upper_name = student.name;
            for (char& c : upper_name) {
                c = std::toupper(c);
            }

            int result_index = -1;
            for (int i = 0; i < num_students; ++i) {
                if (results[i].name.empty()) {
                    result_index = i;
                    break;
                }
            }

            if (result_index >= 0) {
                results[result_index].name = upper_name;
                results[result_index].year_grade = std::to_string(student.year) + grade;
            }
            num_results = atomicAdd(&num_results, 1);
        }
    }
}

__device__ char getGrade(Student student) {
    char grade;
    if (student.grade >= 90) {
        grade = 'A';
    }
    else if (student.grade >= 80) {
        grade = 'B';
    }
    else if (student.grade >= 70) {
        grade = 'C';
    }
    else if (student.grade >= 60) {
        grade = 'D';
    }
    else {
        grade = 'F';
    }
    return grade;
}


void PrintStudentsToFile(Result* studentArray, int numStudents, const std::string& filename) {
    std::ofstream file(filename);

    for (int i = 0; i < numStudents; i++) {
        file << studentArray[i].name << "-" << studentArray[i].year_grade << std::endl;
    }
}


int main() {
    std::ifstream input_file("data1.txt");
    int num_students = 0;
    int num_results = 0;
    std::string line;
    while (std::getline(input_file, line)) {
        ++num_students;
    }
    input_file.clear();
    input_file.seekg(0, std::ios::beg);
    Student* students = new Student[num_students];
    for (int i = 0; i < num_students; ++i) {
        input_file >> students[i];
    }
    input_file.close();

    int num_threads = 64;
    int block_size = 32;
    int num_blocks = (num_students + num_threads - 1) / num_threads;


    Student* d_students;
    hipMalloc(&d_students, num_students * sizeof(Student));
    Result* d_results;
    hipMalloc(&d_results, num_students * sizeof(Result));

    hipMemcpy(d_students, students, num_students * sizeof(Student), hipMemcpyHostToDevice);

    computeResult<<<num_blocks, block_size >>>(d_students, d_results, num_students, num_results);

    Result* results = new Result[num_students];
    hipMemcpy(results, d_results, num_results * sizeof(Result), hipMemcpyDeviceToHost);


    for (int i = 0; i < num_students; ++i) {
        std::cout << results[i].name << "-" << results[i].year_grade << std::endl;
    }

    PrintStudentsToFile(results, num_students, "results.txt");

    delete[] students;
    delete[] results;
    hipFree(d_students);
    hipFree(d_results);

    return 0;
}
